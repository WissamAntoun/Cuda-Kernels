#include "hip/hip_runtime.h"
﻿#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include<opencv2/opencv.hpp>
#include <iostream>
#include <hip/hip_runtime.h>
#include ""
#include "GpuTimer.h"
#include<time.h>

using namespace cv;
using namespace std;


//In OpenCV the image is read in BGR format, that is for each pixel, the Blue, Green, then Red components are read from the image file.

// Serial implementation for running on CPU using a single thread.
void rgbaToGrayscaleCpu(unsigned char* rgbImage, unsigned char* grayImage,int numRows, int numCols, int Channels)
{
	
	//@@ Insert your code here
	for (int y = 0; y < numRows; y++)
	{
		for (int x = 0; x < numCols; x++)
		{
			int grayoff = y*numCols + x;

		int bgroff = grayoff*Channels;

		

		unsigned char Blue = rgbImage[bgroff];
		unsigned char Green = rgbImage[bgroff + 1];
		unsigned char Red = rgbImage[bgroff + 2];

		grayImage[grayoff] = 0.21f*Red + 0.71f*Green + 0.07f*Blue;
		}
	}
}


// we have 3 channels corresponding to B, G, and R components of each pixel
// The input image is encoded as unsigned characters [0, 255]

__global__ void colorToGrayscaleConversion(unsigned char * Pout, unsigned
char * Pin, int width, int height, int numChannels) 
{
	
	//@@ Insert Your Kernel code Here
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < width && y < height)
	{
		int grayoff = y*width + x;
		int bgroff = grayoff*numChannels;	

		unsigned char Blue = Pin[bgroff];
		unsigned char Green = Pin[bgroff + 1];
		unsigned char Red = Pin[bgroff + 2];

		Pout[grayoff] = 0.21f*Red + 0.71f*Green + 0.07f*Blue;
	}
}

__global__ void GPU_inrange(unsigned char * InputImage, int bl, int gl, int rl, int bh, int gh, int rh, unsigned char * Threshold, int numRows, int numCols)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int row = blockIdx.y*blockDim.y + ty;
	int col = blockIdx.x*blockDim.x + tx;



	if ((row < numRows) && (col < numCols))
	{
		unsigned char btemp = InputImage[(row*numCols + col) * 3];
		unsigned char gtemp = InputImage[(row*numCols + col) * 3 + 1];
		unsigned char rtemp = InputImage[(row*numCols + col) * 3 + 2];

		if (btemp > bl && btemp<bh && gtemp>gl && gtemp + 1< gh && rtemp > rl && rtemp< rh)
			Threshold[row*numCols + col] = 255;
		else
			Threshold[row*numCols + col] = 0;
	}

}



int main(void)
{
	//Read the image using OpenCV
	Mat image; //Create matrix to read iamge
	image= imread("scene.jpg",CV_LOAD_IMAGE_COLOR);
	if (image.empty()) {
		printf("Cannot read image file %s", "lena_color.bmp");
		exit(1);
	}

	
	int imageChannels = 3;
	int imageWidth=image.cols;
	int imageHeight=image.rows;

	//Allocate the host image vectors
	unsigned char *h_rgbImage = (unsigned char *)malloc(sizeof(unsigned char)*imageWidth*imageHeight*imageChannels);
	unsigned char *h_grayImage= (unsigned char *)malloc(sizeof(unsigned char)*imageWidth*imageHeight);
	unsigned char *h_grayImage_CPU= (unsigned char *)malloc(sizeof(unsigned char)*imageWidth*imageHeight);

	h_rgbImage = image.data; //The data member of a Mat object returns the pointer to the first row, first column of the image.
							 //try image.ptr()


	//Allocate memory on the device for the rgb image and the grayscale image and record the needed time
	unsigned char *d_rgbImage, *d_grayImage;
	GpuTimer timer;
	timer.Start();
	
	//@@ Insert Your code Here to allocate memory on the device for color and gray images
	hipError_t err;
	//allocating rgbimage
	d_rgbImage = NULL;
	err = hipMalloc((void **)&d_rgbImage, sizeof(unsigned char)*imageWidth*imageHeight*imageChannels);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector rgbImage (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//allocationg grayimage
	d_grayImage = NULL;
	err = hipMalloc((void **)&d_grayImage, sizeof(unsigned char)*imageWidth*imageHeight);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector grayImage (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	

	timer.Stop();
	printf("Time to allocate memory on the device is: %f msecs.\n", timer.Elapsed());

	

	//Copy the rgb image from the host to the device and record the needed time
	GpuTimer timer1;
	timer1.Start();
	
	//@@ Insert your code here to Copy the rgb image from the host to the device
	hipMemcpy(d_rgbImage, h_rgbImage, sizeof(unsigned char)*imageWidth*imageHeight*imageChannels, hipMemcpyHostToDevice);

	timer1.Stop();
	printf("Time to copy the RGB image from the host to the device is: %f msecs.\n", timer1.Elapsed());

	
	//Do the Processing on the GPU
	//Kernel Execution Configuration Parameters
	dim3 dimBlock(16, 16, 1);
	
	//@@ Insert Your code Here for grid dimensions
	dim3 dimGrid((imageWidth - 1) / 16 + 1, (imageHeight - 1) / 16 + 1, 1);
	
	
	//Invoke the colorToGrayscaleConversion kernel and record the needed time for its execution
	//GpuTimer timer;
	GpuTimer timer2;
	timer2.Start();
	int lower_bound[] = { 40,0,90 };
	int higher_bound[] = { 256,95,256 };
	//@@ Insert your code here for kernel invocation
	//colorToGrayscaleConversion <<<dimGrid, dimBlock >>> (d_grayImage, d_rgbImage, imageWidth, imageHeight, 3);
	GPU_inrange << <dimGrid, dimBlock >> >(d_rgbImage, 40, 0, 90, 256, 95, 256, d_grayImage, imageHeight, imageWidth);
	timer2.Stop();
	printf("Implemented CUDA code ran in: %f msecs.\n", timer2.Elapsed());

	//Copy resulting gray image from device to host and record the needed time
	GpuTimer timer3;
	timer3.Start();
	
	//@@ Insert your code here to Copy resulting gray image from device to host 
	hipMemcpy(h_grayImage, d_grayImage, sizeof(unsigned char)*imageWidth*imageHeight, hipMemcpyDeviceToHost);

	timer3.Stop();
	printf("Time to copy the Gray image from the device to the host is: %f msecs.\n", timer3.Elapsed());

	

	//Do the Processing on the CPU
	clock_t begin = clock();
	
	//@@ Insert your code her to call the cpu function for colortograyscale conversion on the CPU
	rgbaToGrayscaleCpu(h_rgbImage, h_grayImage_CPU, imageHeight, imageWidth, 3);

	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC*1000;
	printf("Implemented CPU code ran in: %f msecs.\n", time_spent);

	//Postprocess and Display the resulting images using OpenCV
	Mat Image1(imageHeight, imageWidth,CV_8UC1,h_grayImage); //grayscale image mat object
	Mat Image2(imageHeight,imageWidth,CV_8UC1,h_grayImage_CPU ); //grayscale image mat object

	

	namedWindow("CPUImage", WINDOW_NORMAL); //Create window to display the image
	namedWindow("GPUImage", WINDOW_NORMAL);
	imshow("GPUImage",Image1);
	imshow("CPUImage",Image2); //Display the image in the window
	waitKey(0); //Wait till you press a key 

	
	
	//Free host memory
	image.release();
	Image1.release();
	Image2.release();
	free(h_grayImage);
	free(h_grayImage_CPU);

	//Free device memory
	
	//@@ Insert your code here to free device memory
	hipFree(d_grayImage);
	hipFree(d_rgbImage);

	return 0;

}