#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include<math.h>
#include <stdio.h>
#include<time.h>
#include <iostream>
#include <stdlib.h>
#include "GpuTimer.h"
using namespace std;

#define BLOCK_SIZE 256
#define SegLength 1024



//CUDA Kernel Device code
//Computes the element-wise vector addition of A and B into C: C[i] = A[i] + B[i].
//The 3 vectors have the same number of elements numElements.
__global__ void vectorAdd(float *A, float *B, float *C, int numElements)

{
	//@@ Insert  your code here to implement vector addition
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < numElements)
		C[i] = A[i] + B[i];

}

/**
* Host main routine
*/
int main(void)
{
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Print the vector length to be used, and compute its size
	float EPS = 0.0001;
	int numElements = 1000000;
	size_t size = numElements * sizeof(float);
	printf("[Vector addition of %d elements]\n", numElements);

	//Implement Vector Addition without using CUDA Streams
	// Allocate the host input vector A
	float *h_A = (float *)malloc(size);

	// Allocate the host input vector B
	float *h_B = (float *)malloc(size);

	// Allocate the host output vector C
	float *h_C = (float *)malloc(size);

	// Verify that allocations succeeded
	if (h_A == NULL || h_B == NULL || h_C == NULL)
	{
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// Initialize the host input vectors
	for (int i = 0; i < numElements; ++i)
	{
		h_A[i] = float(i);
		h_B[i] = 1 / (i + EPS);
	}

	// Allocate the device input vector A
	float *d_A = NULL;
	err = hipMalloc((void **)&d_A, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector B
	float *d_B = NULL;
	err = hipMalloc((void **)&d_B, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device output vector C
	float *d_C = NULL;
	err = hipMalloc((void **)&d_C, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the host input vectors A and B in host memory to the device input vectors in
	// device memory
	GpuTimer timer;
	timer.Start();
	//printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	timer.Stop();
	printf("Time to copy the input array A from the host to the device is: %f msecs.\n", timer.Elapsed());


	GpuTimer timer2;
	timer2.Start();

	err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	timer2.Stop();
	printf("Time to copy the input array B from the host to the device is: %f msecs.\n", timer2.Elapsed());


	GpuTimer timer3;
	timer3.Start();
	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	vectorAdd << <blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_C, numElements);
	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	timer3.Stop();
	printf("Implemented CUDA code for vector addition ran in: %f msecs.\n", timer3.Elapsed());


	GpuTimer timer4;
	timer4.Start();
	// Copy the device result vector in device memory to the host result vector
	// in host memory.
	//printf("Copy output data from the CUDA device to the host memory\n");
	err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	timer4.Stop();
	printf("Time to copy the output array C from the Device to the Host is: %f msecs.\n", timer4.Elapsed());


	// Verify that the result vector is correct
	for (int i = 0; i < numElements; ++i)
	{
		if (fabs((h_A[i] + h_B[i]) - h_C[i]) > 1e-5)
		{
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
	}

	printf("Test PASSED\n");

	// Free device global memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);



	//Implement Vector Addition Using CUDA Streams

	//@@ Insert your code here to implement Vector Addition using streams and Time your implementation.

	// Free host memory
	free(h_A);
	free(h_B);
	free(h_C);

	printf("Done\n");

	return 0;
}

